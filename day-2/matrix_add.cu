#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// CUDA-SPECIFIC: Matrix addition kernel using 2D thread blocks
__global__ void matrixAdd(const float *A, const float *B, float *C, int width, int height)
{
    // Calculate global thread indices
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Check bounds
    if (col < width && row < height)
    {
        int index = row * width + col;
        C[index] = A[index] + B[index];
    }
}

int main(void)
{
    // ========== PRODUCTION FEATURE: Dynamic Matrix Size ==========
    // Can handle large matrices, not just small fixed sizes
    int width = 1024;   // Matrix width
    int height = 1024;  // Matrix height
    int numElements = width * height;
    size_t size = numElements * sizeof(float);
    
    printf("[Matrix addition of %dx%d matrices]\n", width, height);
    
    // Allocate host (CPU) memory
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);
    
    // ========== PRODUCTION FEATURE: NULL Check for Host Memory ==========
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host matrices!\n");
        exit(EXIT_FAILURE);
    }
    
    // Initialize matrices with sequential values
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = (float)i;
        h_B[i] = (float)(i * 2);
    }
    
    // Device (GPU) memory pointers
    float *d_A = NULL;
    float *d_B = NULL;
    float *d_C = NULL;
    hipError_t err = hipSuccess;
    
    // ========== CUDA EVENTS FOR PRECISE GPU TIMING ==========
    hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);
    
    // ========== PRODUCTION FEATURE: Error Checking for GPU Allocation ==========
    err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMalloc((void **)&d_B, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMalloc((void **)&d_C, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Copy matrices from host to device
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy matrix A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy matrix B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // ========== PRODUCTION FEATURE: 2D Thread Organization for Matrices ==========
    // Using 2D blocks for better spatial locality and easier indexing
    dim3 threadsPerBlock(16, 16);  // 16x16 = 256 threads per block
    dim3 blocksPerGrid(
        (width + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (height + threadsPerBlock.y - 1) / threadsPerBlock.y
    );
    
    printf("CUDA kernel launch with %dx%d blocks of %dx%d threads\n",
           blocksPerGrid.x, blocksPerGrid.y, threadsPerBlock.x, threadsPerBlock.y);
    
    // ========== TIMING: Start GPU measurement (kernel only) ==========
    hipEventRecord(start_event);
    
    // Launch the Matrix Addition CUDA Kernel
    matrixAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, width, height);
    
    // Check for kernel launch errors
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch matrixAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Wait for kernel to complete
    err = hipDeviceSynchronize();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to synchronize device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // ========== TIMING: End GPU measurement ==========
    hipEventRecord(stop_event);
    hipEventSynchronize(stop_event);
    float gpu_time = 0;
    hipEventElapsedTime(&gpu_time, start_event, stop_event);
    
    // Copy result matrix from device to host
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy matrix C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // ========== PRODUCTION FEATURE: Result Verification ==========
    // Verify that the result is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            fprintf(stderr, "Expected: %f, Got: %f\n", h_A[i] + h_B[i], h_C[i]);
            exit(EXIT_FAILURE);
        }
    }
    
    printf("GPU computation time: %.3f ms (%d elements)\n", gpu_time, numElements);
    printf("✅ Test PASSED - All %d elements computed correctly!\n", numElements);
    
    // Print sample of result (first 4x4 block)
    printf("\nSample of result matrix (top-left 4x4):\n");
    for (int i = 0; i < 4 && i < height; i++)
    {
        for (int j = 0; j < 4 && j < width; j++)
        {
            printf("%6.0f ", h_C[i * width + j]);
        }
        printf("\n");
    }
    
    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    // Clean up CUDA events
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);
    
    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);
    
    printf("\nDone\n");
    return 0;
}