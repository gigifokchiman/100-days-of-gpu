#include <stdio.h>              // Standard C header
#include <hip/hip_runtime.h>       // CUDA-SPECIFIC: CUDA runtime API header

// CUDA-SPECIFIC: __global__ declares a GPU kernel function
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    // CUDA-SPECIFIC: Built-in variables for thread indexing
    // blockDim.x = threads per block, blockIdx.x = block index, threadIdx.x = thread index within block
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

int main(void)
{
    // ========== PRODUCTION FEATURE: Dynamic Array Size ==========
    // Can handle any size, not just small fixed arrays
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);
    
    // Standard C: Allocate host (CPU) memory
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);
    
    // ========== PRODUCTION FEATURE: NULL Check for Host Memory ==========
    // Essential for catching out-of-memory conditions
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }
    
    // Initialize with random values for testing
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;  // Random float between 0.0 and 1.0
        h_B[i] = rand()/(float)RAND_MAX;
    }
    
    // Device (GPU) memory pointers
    float *d_A = NULL;
    float *d_B = NULL;
    float *d_C = NULL;
    // CUDA-SPECIFIC: Error handling type
    hipError_t err = hipSuccess;
    
    // ========== PRODUCTION FEATURE: Error Checking for GPU Allocation ==========
    // CUDA-SPECIFIC: Allocate memory on GPU device
    err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess)
    {
        // CUDA-SPECIFIC: cudaGetErrorString converts error code to readable string
        // Provides detailed error info (e.g., "out of memory", "invalid value")
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMalloc((void **)&d_B, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMalloc((void **)&d_C, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    printf("Copy input data from the host memory to the CUDA device\n");
    // CUDA-SPECIFIC: Copy data from host (CPU) to device (GPU)
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // ========== PRODUCTION FEATURE: Scalable Thread Organization ==========
    // CUDA thread organization for handling arrays of any size
    int threadsPerBlock = 256;  // Common choice for good occupancy
    // Calculate grid size to ensure all elements are processed
    // The +threadsPerBlock-1 ensures we round up (ceiling division)
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    
    // CUDA-SPECIFIC: <<<blocks, threads>>> kernel launch syntax
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    
    // ========== PRODUCTION FEATURE: Kernel Launch Error Detection ==========
    // CUDA-SPECIFIC: Check for kernel launch errors
    // Kernels launch asynchronously, so errors might not appear immediately
    err = hipGetLastError();
    
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    printf("Copy output data from the CUDA device to the host memory\n");
    // CUDA-SPECIFIC: Copy results from device (GPU) to host (CPU)
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // ========== PRODUCTION FEATURE: Result Verification ==========
    // Validate that GPU computation matches expected results
    // Essential for catching numerical errors or kernel bugs
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)  // Floating point tolerance
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    
    printf("Test PASSED\n");
    
    // CUDA-SPECIFIC: Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    // Standard C: Free CPU memory
    free(h_A);
    free(h_B);
    free(h_C);
    
    printf("Done\n");
    return 0;
}